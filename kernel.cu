//#include "kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"


#define N 5000

__host__
bool checkArr(int *arr, int size)
{
	for (int i = 0; i < size-1; ++i)
	{
		if (arr[i] > arr[i + 1])
		{
			printf("Array index: %d, with value: %d\nIs greater than index: %d, with value: %d\n", i, arr[i], i + 1, arr[i + 1]);
		}
	}
	return true;
}

__host__
void printArr(int *arr, int size)
{
	for (int i = 0; i < size - 1; ++i)
	{
		printf("%d, ", arr[i]);
	}
	printf("%d \n\n", arr[size - 1]);
}

__host__
void createRandArr(int *arr, int size, int maxVal)
{
	for (int i = 0; i < size; ++i)
		arr[i] = (rand() / (float)(RAND_MAX)) * maxVal;
}

__host__
int oddeven(int *arr, int size, int oddeven)
{
	int sorted = 0;
	for (int i = oddeven; i < size-oddeven; i += 2)
	{
		int minStep = arr[i] > arr[i + 1];
		int min = arr[i + minStep];
		int maxStep = arr[i] <= arr[i + 1];
		int max = arr[i + maxStep];

		arr[i] = min;
		arr[i + 1] = max;

		sorted += minStep - maxStep;
	}
	return sorted;
}

__host__
void sortCPU(int *arr, int size)
{
	int i = 0;
	int sorted = 1;
	while (sorted != (-size+1))
	{
		sorted = oddeven(arr, size, i % 2);
		sorted += oddeven(arr, size, (i+1) % 2);
		i += 2;
	}
}

//__device__
//int oddevenGPU(int *d_arr, int size, int oddeven, int blockSize, int startIndex, int endIndex)
//{
//	int sorted = 0;
//	for (int i = startIndex; i < endIndex; i += 2)
//	{
//		int minStep = d_arr[i] > d_arr[i + 1];
//		int min = d_arr[i + minStep];
//		int maxStep = d_arr[i] <= d_arr[i + 1];
//		int max = d_arr[i + maxStep];
//
//		d_arr[i] = min;
//		d_arr[i + 1] = max;
//
//		sorted += minStep - maxStep;
//	}
//	return sorted;
//}
//
//__global__
//void addKernel(int *d_arr, int *d_size, int *d_blockSize, int *d_sorted)
//{
//	int size = *d_size;
//	int blockSize = *d_blockSize;
//	int nrThreads = size / blockSize;
//	int elemInThread = size / nrThreads;
//	int shift = elemInThread % 2;
//
//	int i = 0;
//	int sorted = 0;
//	int oddeven = 0;
//	while (sorted != (-size + 1))
//	{
//		sorted = 0;
//
//
//		oddeven = i % 2;	//0 == odd, 1 == even
//		int startIndex = blockSize * threadIdx.x + oddeven + (shift * ((threadIdx.x + 1) % 2) * threadIdx.x != 0);
//		int endIndex = blockSize + blockSize * threadIdx.x - oddeven + shift * ((threadIdx.x + 1)%2);
//
//		sorted += oddevenGPU(d_arr, size, oddeven, blockSize, startIndex, endIndex);
//		__syncthreads();
//
//
//		oddeven = (i + 1) % 2;
//		startIndex = blockSize * threadIdx.x + oddeven;
//		endIndex = blockSize + blockSize * threadIdx.x - oddeven;
//
//		sorted += oddevenGPU(d_arr, size, oddeven, blockSize, startIndex, endIndex);
//		__syncthreads();
//		i += 2;
//	}
//}

//int main()
//{
//	srand((unsigned int)time(NULL));
//
//	int size = 100;
//	int *arr = (int*)malloc(size * sizeof(int));
//	int *d_arr, *d_size, *d_blockSize, *d_sorted;
//	createRandArr(arr, size, size*2);
//
//	int n = 2;
//	int blockSize = size / n;
//
//	printArr(arr, size);
//	
//	cudaMalloc(&d_arr, size * sizeof(int));
//	cudaMalloc(&d_size, sizeof(int));
//	cudaMalloc(&d_blockSize, sizeof(int));
//	cudaMalloc(&d_sorted, sizeof(int));
//
//	cudaMemcpy(d_arr, arr, size * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_size, &size, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_blockSize, &blockSize, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_sorted, &size, sizeof(int), cudaMemcpyHostToDevice);
//
//	int nr = size / blockSize;
//	
//	addKernel<<<1, (size/2)>>>(d_arr, d_size, d_blockSize, d_sorted);
//	cudaMemcpy(arr, d_arr, size * sizeof(int), cudaMemcpyDeviceToHost);
//
//	printArr(arr, size);
//
//	/*printArr(arr, size);
//	sortCPU(arr, size);
//	printArr(arr, size);*/
//
//	system("pause");
//
//	cudaFree(d_arr);
//	cudaFree(d_size);
//	free(arr);
//	return 0;
//}

__global__
void oddeven(int *arr, int flag, int nrThreads, int size)
{
	int d_flag = flag%2;
	int sizeNR = (size / nrThreads) + ((size / nrThreads) % 2);
	int index = (blockIdx.x * blockDim.x + threadIdx.x) * sizeNR;
	if ((index >= size - 1) && d_flag != 0) return;	//Out of bounds

	int end = sizeNR + (blockIdx.x * blockDim.x + threadIdx.x) * sizeNR;

	if (end >= size -1 - d_flag)
		end = size - 1 - d_flag;

	index += d_flag;

	for (int i = index; i < end; i += 2)
	{
		int min = arr[i + (arr[i] > arr[i + 1])];
		int max = arr[i + (arr[i] <= arr[i + 1])];

		arr[i] = min;
		arr[i + 1] = max;
	}
}

int main()
{
	int *arr;
	int *d_arr;
	int i;
	int size = sizeof(int) * N;
	srand((unsigned)time(NULL));

	arr = (int*)malloc(size);

	hipMalloc(&d_arr, size);

	createRandArr(arr, N, N * 2);

	//printArr(arr, N);

	

	hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);


	double start_time = clock();
	for (i = 0; i < N; ++i)
	{
		oddeven<<<1, 500>>>(d_arr, i, 500, N);
	}
	printf("\nExecution time: %lf seconds.\n", (clock() - start_time) / CLOCKS_PER_SEC);
	hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);

	

	//printArr(arr, N);

	bool sorted = checkArr(arr, N);

	system("pause");

	return 0;
}